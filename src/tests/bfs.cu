#define BFS_MEM_H
#define BFS_VAL_TYPE
#include <stdio.h>
#include <stdio.h>
#include <string>
#include "../preprocessing/PreProcessing.hpp"
#include "../gpuengine/excuteGPU.h"
#include "../preprocessing/prepartitionGPU.hpp"

using namespace std;

int main(int argc, char *argv[])
{
	if(argc < 3)
	{
		std::cout << "main input error." << std::endl;
		exit(-1);
	}

	std::string file_name = argv[1];
	int chunk = atoi(argv[2]);

	PreProcessing *pre = new PreProcessing;

	pre->get_partition(file_name, chunk);
	
	set_n_edge(chunk, 0);
	unsigned int e_num = pre->get_all_edges(file_name, chunk);
	unsigned int v_num = pre->vv.get_vertex_num();

	for(int j = 0; j < chunk; j++)
        {
                std::cout<< "n_edge[ " << j << " ] = " << num[j] << std::endl;
        }

	init_cuda_stream(chunk);
        for (int i = 0; i < chunk; i++)
        {
                hipStreamCreate(&stream[i]);
        }

	pre->get_n_vertex_partition(chunk);
	memcpy_vertex_m2d(chunk, 0);

    std::cout << "vv.size = " << pre->vv.vv.size();
    {
         std::set<unsigned int> temp;
         temp.swap(pre->vv.vv);
         if(pre->vv.vvArray != NULL)
         {
              free(pre->vv.vvArray);
         }
    }
    std::cout << ", free vertexvector size = " << pre->vv.vv.size() << ", vvArray = " << sizeof(pre->vv.vvArray) << std::endl;

	//mem edges for each chunk
	mem_all_edges(chunk);
	std::cout << "mem_all_edges ... " <<std::endl;
	pre->set_partition_edges_src(chunk, e_num);
    printf("set_partition_edges_src success......\n");

    d_mem_max_edges(chunk);

	mem_all_vertex(chunk);
	set_partition_vertex_src(chunk);
	d_mem_all_vertex(chunk);

    std::cout << "deleting preprocessing in main ..." << std::endl;
    delete pre;

	hipEvent_t start, stop, start1, stop1, start2, stop2;
	float gpu_time_excute = 0, gpu_time = 0, gpu_time_edge = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&start1);
        hipEventCreate(&stop1);
	hipEventCreate(&start2);
        hipEventCreate(&stop2);

	//values mem
	mem_vertex_values(v_num);
        for(unsigned int v = 0; v < m_size; v++)
        {
                m_value[v] = INIT_VAL;
        }
        m_value[ROOT_ID] = 0;
	d_mem_vertex_values(v_num);
	memcpy_value_m2d(0);

	std::cout << "Starting GPU processing and timeing...." << std::endl;

	int *m_stop = (int *)malloc(sizeof(int) * 1), *d_stop;
	hipMalloc((void **)&d_stop, sizeof(int) * 1);
	m_stop[0] = 1;
	hipEventRecord(start1, 0);
	//for(int i = 0; i < iter && ; i++)
	int iter = 30, i = 0, j;
	while(m_stop[0])
	{
		m_stop[0] = 0;
		//hipMemcpy(d_stop, m_stop, sizeof(int) * 1, hipMemcpyHostToDevice);
		hipMemset(d_stop, 0, sizeof(int));
		//int j;
		for(j = 0; j < chunk; j++)
		{
			int blockv = (n_vertex[j] * W_SE + 256 - 1) / 256;
            		memcpy_chunk_edges_m2d(j);
			BFS_E<<<blockv, 256>>>(d_value, d_max_edge, num[j], i, d_stop);
		}
		i++;
		hipMemcpy(m_stop, d_stop, sizeof(int) * 1, hipMemcpyDeviceToHost);
	}

	hipEventRecord(stop1, 0);
	hipEventSynchronize(stop1);
	hipEventElapsedTime(&gpu_time, start1, stop1);

	std::cout <<"Total GPU proc time is: " << gpu_time << std::endl;

	memcpy_value_d2m(0);

	for (int i = 0; i < chunk; i++)
        {
                hipStreamSynchronize(stream[i]);
        }
        for (int i = 0; i < chunk; i++)
        {
                hipStreamDestroy(stream[i]);
        }
	
	unsigned int step = 0, reached = 0;
	for(unsigned int v = 0; v < v_num; v++)
	{
		if(m_value[v] > step && m_value[v] != INIT_VAL)
                {
                        step = m_value[v];
                }

		if(v < 10)
		{
			std::cout << v <<" vertex pagerank value is " << m_value[v] << std::endl;
		}
	
	}

	std::cout << "Result is step = " << step << " , Total reached edges = " << reached << std::endl;

	release_n_edge();
	release_mem_values();
    d_release_mem_values();
	release_all_edges();
	release_d_max_edges();

	std::cout << "......Preprossing Complete !" << std::endl;

	return 1;
}
