#define CC_MEM_H
#define CC_VAL_TYPE
#include <stdio.h>
#include <stdio.h>
#include <string>
#include "../preprocessing/PreProcessing.hpp"
#include "../gpuengine/excuteGPU.h"
#include "../preprocessing/prepartitionGPU.hpp"

using namespace std;

int main(int argc, char *argv[])
{
	if(argc < 3)
	{
		std::cout << "main input error." << std::endl;
		exit(-1);
	}

	std::string file_name = argv[1];
	int chunk = atoi(argv[2]);

	PreProcessing *pre = new PreProcessing;

	pre->get_partition(file_name, chunk);
	
	set_n_edge(chunk, 0);
	unsigned int e_num = pre->get_all_edges(file_name, chunk);
	unsigned int v_num = pre->vv.get_vertex_num();

	for(int j = 0; j < chunk; j++)
        {
                std::cout<< "n_edge[ " << j << " ] = " << num[j] << std::endl;
        }

	init_cuda_stream(chunk);
        for (int i = 0; i < chunk; i++)
        {
                hipStreamCreate(&stream[i]);
        }

	pre->get_n_vertex_partition(chunk);
	memcpy_vertex_m2d(chunk, 0);

    std::cout << "vv.size = " << pre->vv.vv.size();
    {
         std::set<unsigned int> temp;
         temp.swap(pre->vv.vv);
         if(pre->vv.vvArray != NULL)
         {
              free(pre->vv.vvArray);
         }
    }
    std::cout << ", free vertexvector size = " << pre->vv.vv.size() << ", vvArray = " << sizeof(pre->vv.vvArray) << std::endl;

	//mem edges for each chunk
	mem_all_edges(chunk);
	std::cout << "mem_all_edges ... " <<std::endl;
	pre->set_partition_edges_src(chunk, e_num);
    printf("set_partition_edges_src success......\n");

    d_mem_max_edges(chunk);

	mem_all_vertex(chunk);
	set_partition_vertex_src(chunk);
	d_mem_all_vertex(chunk);

    std::cout << "deleting preprocessing in main ..." << std::endl;
    delete pre;

	hipEvent_t start, stop, start1, stop1, start2, stop2, start_cc, stop_cc;
	float gpu_time_excute = 0, gpu_time = 0, gpu_time_edge = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&start1);
        hipEventCreate(&stop1);
	hipEventCreate(&start2);
        hipEventCreate(&stop2);
	hipEventCreate(&start_cc);
        hipEventCreate(&stop_cc);

	//values mem
	mem_vertex_values(v_num);
        for(unsigned int v = 0; v < m_size; v++)
        {
                m_value[v] = v;
        }
        //m_value[ROOT_ID] = 0;
	d_mem_vertex_values(v_num);
	memcpy_value_m2d(0);

	for(int i = 0; i < chunk; i++)
	{
		memcpy_all_vertex_m2d(i, i);
	}

	std::cout << "Starting GPU processing and timeing...." << std::endl;

	int *m_stop = (int *)malloc(sizeof(int) * 1), *d_stop;
	hipMalloc((void **)&d_stop, sizeof(int) * 1);
	m_stop[0] = 1;
	hipEventRecord(start1, 0);
	//for(int i = 0; i < iter && ; i++)
	int iter = 30, i = 0, j;
	while(m_stop[0])
	{
		m_stop[0] = 0;
		hipMemcpy(d_stop, m_stop, sizeof(int) * 1, hipMemcpyHostToDevice);
		//hipMemset(d_stop, 0, sizeof(int));
		//int j;
		for(j = 0; j < chunk; j++)
		{
			int blockv = (num[j] + 256 - 1) / 256;
            		memcpy_chunk_edges_m2d(j);
			hipEventRecord(start_cc, 0);
			Component_E<<<blockv, 256>>>(d_value, d_max_edge, num[j], 0, d_stop);
			hipDeviceSynchronize();
                        hipEventRecord(stop_cc, 0);
                        hipEventSynchronize(stop_cc);
                        float tmp_time;
                        hipEventElapsedTime(&tmp_time, start_cc, stop_cc);
                        gpu_time_excute += tmp_time;
		}
		i++;
		hipMemcpy(m_stop, d_stop, sizeof(int) * 1, hipMemcpyDeviceToHost);
	}

	hipEventRecord(stop1, 0);
	hipEventSynchronize(stop1);
	hipEventElapsedTime(&gpu_time, start1, stop1);

	std::cout <<"Total GPU proc time is: " << gpu_time_excute << std::endl;

	memcpy_value_d2m(0);

	for (int i = 0; i < chunk; i++)
        {
                hipStreamSynchronize(stream[i]);
        }
        for (int i = 0; i < chunk; i++)
        {
                hipStreamDestroy(stream[i]);
        }
	
	std::set<unsigned int> result;
        for(unsigned int v = 0; v < m_size; v++)
        {
                result.insert(m_value[v]);
	}

	unsigned int *l_size, tmp_size = result.size();
        l_size = (unsigned int *) malloc(sizeof(unsigned int) * tmp_size);
        memset(l_size, 0, sizeof(unsigned int) / sizeof(char) * tmp_size);

        for(unsigned int v = 0; v < v_num; v++)
        {
                l_size[m_value[v] % tmp_size]++;
        }

        for(unsigned int i = 0; i < 20; i++)
        {
                std::cout << "level " << i << " has vertex " << l_size[i] << std::endl;
		std::cout << "value " << i << " is " << m_value[i] << std::endl;
        }
        free(l_size);
        std::cout << "Result is component = " << tmp_size << std::endl;

	release_n_edge();
	release_mem_values();
    	d_release_mem_values();
	release_all_edges();
	release_d_max_edges();

	std::cout << "......Preprossing Complete !" << std::endl;

	return 1;
}
